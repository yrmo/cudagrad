// Copyright 2023-2024 Ryan Moore


#include <hip/hip_runtime.h>
#include <stdio.h>

#include <cassert>
#include <string>

__global__ void foo() {}

int main() {
  foo<<<1, 1>>>();
  assert(std::string(hipGetErrorString(hipGetLastError())) == "no error");
}
